#include "cuda_linker.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "linker.hpp"
#include "poses.hpp"
#include "geom3d/quaternion.hpp"
#include "score/score.hpp"
#include "pdbreader/nrset.hpp"
#include "pdbreader/bond.hpp"
#include "helper/benchmark.hpp"
#include "helper/help.hpp"
#include "helper/array2d.hpp"
#include "graph/mcqd.hpp"
#include "modeler/modeler.hpp"
#include "geom3d/geom3d.hpp"
#include "cluster/greedy.hpp"
#include "partial.hpp"
#include <queue>
#include <iostream>

namespace Linker{



void cuda_linker::setup(const int segment_graph_size, const Partial &start_conformation, vector<unique_ptr<State>> &states, int iter){
    //First load all data onto gpu



}
}
