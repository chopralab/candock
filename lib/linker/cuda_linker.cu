#include "hip/hip_runtime.h"
#include "cuda_linker.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "linker.hpp"
#include "poses.hpp"
#include "geom3d/quaternion.hpp"
#include "score/score.hpp"
#include "pdbreader/nrset.hpp"
#include "pdbreader/bond.hpp"
#include "helper/benchmark.hpp"
#include "helper/help.hpp"
#include "helper/array2d.hpp"
#include "graph/mcqd.hpp"
#include "modeler/modeler.hpp"
#include "geom3d/geom3d.hpp"
#include "cluster/greedy.hpp"
#include <queue>
#include <iostream>
__global__ void find_good_neighbor() {

}



void cuda_linker::setup() {
    


}
