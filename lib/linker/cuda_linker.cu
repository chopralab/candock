#include "cuda_linker.h"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"

#include "linker.hpp"
#include "poses.hpp"
#include "geom3d/quaternion.hpp"
#include "score/score.hpp"
#include "pdbreader/nrset.hpp"
#include "pdbreader/bond.hpp"
#include "helper/benchmark.hpp"
#include "helper/help.hpp"
#include "helper/array2d.hpp"
#include "graph/mcqd.hpp"
#include "modeler/modeler.hpp"
#include "geom3d/geom3d.hpp"
#include "cluster/greedy.hpp"
#include "partial.hpp"
#include <queue>
#include <iostream>

namespace Linker{



void cuda_linker::setup(const int segment_graph_size, vector<unique_ptr<State>> &states, int iter, int num_states, int num_docked_seeds){
    //First load all data onto gpu

    //Partial *dev_start_conformation;
    //vector<unique_ptr<State>> *dev_states;
    //int dev_segment_graph_size, dev_inter;
    //hipMalloc(&dev_states, sizeof(states) * num_states);
//    hipMalloc(&dev_start_conformation, sizeof());
    //hipMemcpy();
    
    
//    hipFree(dev_states);

}
}
