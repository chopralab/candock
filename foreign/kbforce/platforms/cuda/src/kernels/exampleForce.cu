#include "hip/hip_runtime.h"
extern "C" __global__ void calcNonBonded(real4* __restrict__ posq, unsigned long long* __restrict__ forceBuffers, int *particle1, int *particle2, int *dev_type1, int *dev_type2, double *dev_potential_lookup, double *dev_derivative_lookup, double *dev_energy) {

    for (int atom = blockIdx.x*blockDim.x+threadIdx.x; atom < num_bonds; atom += blockDim.x*gridDim.x) {
       
      
        
        int p1 = particle1[atom];
        int p2 = particle2[atom];
        
        real4 pos1 = posq[p1];
        real4 pos2 = posq[p2];
        
        
   
        real3 delta = make_real3(pos2.x-pos1.x, pos2.y-pos1.y, pos2.z-pos1.z);
        real r = SQRT(delta.x*delta.x + delta.y*delta.y + delta.z*delta.z);
        size_t dist = static_cast<size_t>(floor(r / step));
      
        
        const int internal_min = (dev_type1[atom] <= dev_type2[atom]) ? dev_type1[atom] : dev_type2[atom];
        const int internal_max = (dev_type1[atom] <= dev_type2[atom]) ? dev_type2[atom] : dev_type1[atom];
    
        const int offset = (internal_max + internal_min*(number_of_types-1)-internal_min*(internal_min-1)/2) * number_of_steps;

        if (dist >= number_of_steps) 
            continue; // effectively add zero to energy and force
            
        atomicAdd(&dev_energy[0], dev_potential_lookup[offset + dist]);
         
         
        real dEdR = dev_derivative_lookup[offset + dist];
        dEdR = (r > 0) ? (dEdR/r) : 0.0;
        
        real3 force1 = make_real3(delta.x * dEdR, delta.y * dEdR, delta.z * dEdR);
        real3 force2 = make_real3(-delta.x * dEdR, -delta.y * dEdR, -delta.z * dEdR);
        
        
        atomicAdd(&forceBuffers[p1], static_cast<unsigned long long>((long long) (force1.x *0x100000000)));
        atomicAdd(&forceBuffers[p2], static_cast<unsigned long long>((long long) (force2.x *0x100000000)));
        
        atomicAdd(&forceBuffers[p1 + PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force1.y *0x100000000)));
        atomicAdd(&forceBuffers[p2 + PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long) (force2.y *0x100000000)));
        
        atomicAdd(&forceBuffers[p1 + PADDED_NUM_ATOMS*2], static_cast<unsigned long long>((long long) (force1.z *0x100000000)));
        atomicAdd(&forceBuffers[p2 + PADDED_NUM_ATOMS*2], static_cast<unsigned long long>((long long) (force2.z *0x100000000)));
    
         
    }
}
    
