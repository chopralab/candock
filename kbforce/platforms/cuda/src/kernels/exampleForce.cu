#include "hip/hip_runtime.h"
{
    real3 delta = posq2 - posq1;
    //real r = sqrt(delta.x * delta.x + delta.y * delta.y + delta.z * delta.z);
    int dist = static_cast<int>(floor(r / step));

    //const int internal_min = dev_atoms[atom1] <= dev_atoms[atom2] ? dev_atoms[atom1] : dev_atoms[atom2];
    //const int internal_max = dev_atoms[atom1] <= dev_atoms[atom2] ? dev_atoms[atom2] : dev_atoms[atom1];
    const int internal_min = atom1 <= atom2 ? atom1 : atom2;
    const int internal_max = atom1 <= atom2 ? atom2 : atom1;


    const int offset = (internal_max + internal_min * (number_of_types - 1) - internal_min * (internal_min - 1) / 2) * number_of_steps;
            
           

    if (dist >= number_of_steps)
        continue; // effectively add zero to energy and force
            

    tempEnergy += dev_potential_lookup[offset + dist];

    dEdR += (r > 0) ? (dev_derivative_lookup[offset + dist] / r) : 0.0;


    real3 force1 = make_real3(delta.x * dEdR, delta.y * dEdR, delta.z * dEdR);
    real3 force2 = make_real3(-delta.x * dEdR, -delta.y * dEdR, -delta.z * dEdR);

    atomicAdd(&forceBuffers[atom1], static_cast<unsigned long long>((long long)(force1.x * 0x100000000)));
    atomicAdd(&forceBuffers[atom2], static_cast<unsigned long long>((long long)(force2.x * 0x100000000)));

    atomicAdd(&forceBuffers[atom1 + PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long)(force1.y * 0x100000000)));
    atomicAdd(&forceBuffers[atom2 + PADDED_NUM_ATOMS], static_cast<unsigned long long>((long long)(force2.y * 0x100000000)));

    atomicAdd(&forceBuffers[atom1 + PADDED_NUM_ATOMS * 2], static_cast<unsigned long long>((long long)(force1.z * 0x100000000)));
    atomicAdd(&forceBuffers[atom2 + PADDED_NUM_ATOMS * 2], static_cast<unsigned long long>((long long)(force2.z * 0x100000000)));
            
   

}


