
#include <hip/hip_runtime.h>
/*extern "C" __global__ void calcNeighborList(
    real4 *__restrict__ posq,
    int2 *dev_neighbors,
    bool *dev_bonded_exclusions_matrix)
{
    int block = blockIdx.x;
    int thread = threadIdx.x;
    printf("block %d thread %d \n", block, thread);
    size_t counter = 0;

    for (size_t interaction = block + 1; interaction < num_atoms; interaction++)
    {

        int atom1 = block;
        int atom2 = interaction;

        if (atom1 > atom2)
        {
            real d_sq = pow(posq[atom1].x - posq[atom2].x, 2) + pow(posq[atom1].y - posq[atom2].y, 2) + pow(posq[atom1].z - posq[atom2].z, 2);

            printf("atom1 %d atom2 %d\n", atom1, atom2);
            if (d_sq < dist_sq && dev_bonded_exclusions_matrix[atom1 * num_atoms + atom2] == false)
            {
                if (counter >= 100)
                    printf("counter is greater than 100!!!\n");

                dev_neighbors[atom1 * 100 + counter++] = make_int2(atom1, atom2);
            }
        }
    }
*/
    /* 
     * We want to make sure any spots not used will not be confused for atoms
     * therefore we set them to -1, because that is not a valid atom number.
     */

  /*  while(counter < num_atoms)
        dev_neighbors[block * 100 + counter++] = make_int2(-1, -1);
    
}
*/