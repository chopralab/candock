#include "hip/hip_runtime.h"
extern "C" __global__ void reduceEnergy(double* __restrict__ dev_energy, double* __restrict__ dev_output_energy) {
    extern __shared__ mixed tempBuffer[];
    const unsigned int thread = threadIdx.x;
    mixed sum = 0;
    for (unsigned int index = thread; index < bufferSize; index += blockDim.x)
        sum += dev_energy[index];
    tempBuffer[thread] = sum;
    for (int i = 1; i < workGroupSize; i *= 2) {
        __syncthreads();
        if (thread%(i*2) == 0 && thread+i < workGroupSize)
            tempBuffer[thread] += tempBuffer[thread+i];
    }
    if (thread == 0)
        *dev_output_energy = tempBuffer[0];
}











